#include "hip/hip_runtime.h"
%%cu
#include <iostream>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>

void make_filter(float* filter, const int fSize, const float sigma) {
    int center = (fSize)/2;
    double s = 2 * sigma;
    double sum = 0.0;
    for (int i = -center; i <= center; ++i) {
        for (int j = -center; j <= center; ++j) {
            double r = sqrt(i * i + j * j);
            auto res = (exp(-(r * r) / s)) / (M_PI * s);
            filter[(i + center) * fSize + (j + center)] = res;
            sum += res;
        }
    }
}

void print_channels(FILE *fo, char name, const float *ch, int Nrows, int Ncols) {
    fprintf(fo, "%c\n", name);
    for (size_t i = 0; i < Nrows; ++i) {
        for (size_t j = 0; j < Ncols; ++j) {
            int idx = i * Ncols + j;
            fprintf(fo, "%f ", ch[idx]);
        }
        fprintf(fo, "\n");
    }
    fprintf(fo, "\n");
}

__global__ void gaussian_blur(const float *inputChannel, float* blurredChannel, int Nrows, int Ncols,
                              const float* const filter, const int fSize) {

    int column = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (column >= Ncols || row >= Nrows) {
        return;
    }

    float c = 0.0f;

    for (int fCol = 0; fCol < fSize; fCol++) {
        for (int fRow = 0; fRow < fSize; fRow++) {

            int imageCol = column + fCol - fSize / 2;
            int imageRow = row + fRow - fSize / 2;

            imageCol = min(max(imageCol,0),Ncols-1);
            imageRow = min(max(imageRow,0),Nrows-1);

            c += (filter[fRow * fSize + fCol] * inputChannel[imageRow*Ncols+imageCol]);
        }
    }

    blurredChannel[row * Ncols + column] = c;
}

void worker(FILE *fo, float sigma) {
    FILE *file;
    file = fopen("case.txt", "r");

    int Nrows, Ncols;
    if (fscanf(file, "%d%d", &Nrows, &Ncols) == EOF) {
        return;
    }

    int Npixels = (int)Nrows * (int)Ncols;
    size_t size = Npixels * (sizeof(float));

    float *h_r_in = (float *)malloc(size);
    float *h_g_in = (float *)malloc(size);
    float *h_b_in = (float *)malloc(size);

    // сделать в функции, учитывая указатели
    for (size_t i = 0; i < Nrows; ++i) {
      for (size_t j = 0; j < Ncols; ++j) {
         fscanf(file, "%f", &h_r_in[i * Ncols + j]);
      }
    }

    for (size_t i = 0; i < Nrows; ++i) {
      for (size_t j = 0; j < Ncols; ++j) {
         fscanf(file, "%f", &h_g_in[i * Ncols + j]);
      }
    }

    for (size_t i = 0; i < Nrows; ++i) {
      for (size_t j = 0; j < Ncols; ++j) {
         fscanf(file, "%f", &h_b_in[i * Ncols + j]);
      }
    }

    fclose(file);

    int fSize = 25;
    size_t size_filter = fSize * fSize * sizeof(float);

    printf("SIGMA = %f;\t", sigma);
    float *h_filter = (float *)malloc(size_filter);
    sigma *= sigma;
    make_filter(h_filter, fSize, sigma);

    float *d_r_in;
    hipMalloc(&d_r_in, size);
    hipMemcpy(d_r_in, h_r_in, size, hipMemcpyHostToDevice);

    float *d_g_in;
    hipMalloc(&d_g_in, size);
    hipMemcpy(d_g_in, h_g_in, size, hipMemcpyHostToDevice);

    float *d_b_in;
    hipMalloc(&d_b_in, size);
    hipMemcpy(d_b_in, h_b_in, size, hipMemcpyHostToDevice);

    float *d_filter;
    hipMalloc(&d_filter, size_filter);
    hipMemcpy(d_filter, h_filter, size_filter, hipMemcpyHostToDevice);

    float *d_r_out;
    hipMalloc(&d_r_out, size);
    float *d_g_out;
    hipMalloc(&d_g_out, size);
    float *d_b_out;
    hipMalloc(&d_b_out, size);

    const dim3 blockSize(1, 1, 1);
    const dim3 gridSize(Ncols / blockSize.x + 1, Nrows / blockSize.y + 1, 1);

    float elapsed=0;
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    gaussian_blur<<<gridSize, blockSize>>>(d_r_in, d_r_out, Nrows, Ncols, d_filter, fSize);
    gaussian_blur<<<gridSize, blockSize>>>(d_g_in, d_g_out, Nrows, Ncols, d_filter, fSize);
    gaussian_blur<<<gridSize, blockSize>>>(d_b_in, d_b_out, Nrows, Ncols, d_filter, fSize);

    hipEventRecord(stop, 0);
    hipEventSynchronize (stop);

    hipEventElapsedTime(&elapsed, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Elapsed time %.2f ms\n", elapsed);

    float *h_r_out = (float *)malloc(size);
    hipMemcpy(h_r_out, d_r_out, size, hipMemcpyDeviceToHost);
    float *h_g_out = (float *)malloc(size);
    hipMemcpy(h_g_out, d_g_out, size, hipMemcpyDeviceToHost);
    float *h_b_out = (float *)malloc(size);
    hipMemcpy(h_b_out, d_b_out, size, hipMemcpyDeviceToHost);

    print_channels(fo, 'R', h_r_out, Nrows, Ncols);
    print_channels(fo, 'G', h_g_out, Nrows, Ncols);
    print_channels(fo, 'B', h_b_out, Nrows, Ncols);

    hipFree(d_filter);
    free(h_filter);

    hipFree(d_r_in);
    hipFree(d_r_out);
    hipFree(h_r_out);
    free(h_r_out);

    hipFree(d_g_in);
    hipFree(d_g_out);
    hipFree(h_g_out);
    free(h_g_out);

    hipFree(d_b_in);
    hipFree(d_b_out);
    hipFree(h_b_out);
    free(h_b_out);

    free(h_r_in);
    free(h_g_in);
    free(h_b_in);
}

int main() {
    
    FILE *fo;
    fo = fopen("tree_25_4_1.txt", "w");

    worker(fo, 4);
    fclose(fo);

    return 0;
}
